#include "hip/hip_runtime.h"
#include	"JP/CUDA/JPCuda.h"
using namespace std;

using namespace nvcuda;

#define	D	1024

template < typename F > __global__ void
DOT( F* all, F* query, F* result, size_t W ) {
	auto x = size_t( blockIdx.x * blockDim.x + threadIdx.x );
	if ( x < W ) {
		result[ x ] = 0;
		auto _ = all + D * x;
		for ( int i = 0; i < D; i++ ) result[ x ] += query[ i ] * _[ i ]; 
	}
}

#include	<chrono>
using namespace chrono;

const size_t
MTPB	= CudaDeviceProp().maxThreadsPerBlock;

template < typename F > void
Main( size_t W ) {

	cerr << "numData : " << W / ( 1024 * 1024 ) << "MiB" << endl;
	cerr << "memory  : " << W * sizeof( F ) * D / ( 1024 * 1024 * 1024 ) << "GiB" << endl;

	system_clock::time_point start;
	start = system_clock::now();

	CUDAMemory< F > data( W * D );
	DummyData( data  );
//data.DtoH();
//for ( size_t _ = 0; _ < W * D; _ += 1024 * 8 * D ) cerr << _ << ':' << (float)data( _ ) << endl;

	CUDAMemory< F > result( W );

	cerr << "Initialize: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

	start = system_clock::now();
	DOT<<< ( W + MTPB ) / MTPB, MTPB >>>( data.$, data.$, result.$, W );
	result.DtoH(); //	hipDeviceSynchronize();
	cerr << "duration first: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

	start = system_clock::now();
	DOT<<< ( W + MTPB ) / MTPB, MTPB >>>( data.$, data.$, result.$, W );
	result.DtoH(); //	hipDeviceSynchronize();
	cerr << "duration second: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

	cerr << "STARING VALIDATION" << endl;
	start = system_clock::now();
	data.DtoH();
	auto p = data.Host();
	for ( size_t _ = 0; _ < W; _++ ) {
		auto q = p + D * _;
		double	$ = 0;
		for ( size_t i = 0; i < D; i++ ) $ += double( p[ i ] ) * double( q[ i ] );
		if ( abs( $ - (double)result( _ ) ) > 0.001 ) cout << _ << ':' << $ << ':' <<  (double)result( _ ) << endl;
	}
	cerr << "validation: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;
}

int
main( int argc, char* argv[] ) {
	try {
		Main< FLOAT_TYPE >( size_t( 1024 * 1024 *  2 ) );
	} catch ( char* ex ) {
		cerr << ex << endl;
	}
}
