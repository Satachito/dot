#include "hip/hip_runtime.h"
#include	"JP/CUDA/JPCuda.h"
using namespace std;

using namespace nvcuda;

#define	D	1024

__constant__ FLOAT_TYPE
query[ D ];

__global__ void
DOT( FLOAT_TYPE* all, FLOAT_TYPE* result, size_t W ) {
	auto x = size_t( blockIdx.x * blockDim.x + threadIdx.x );
	if ( x < W ) {
		result[ x ] = 0;
		auto _ = all + D * x;
		for ( int i = 0; i < D; i++ ) result[ x ] += query[ i ] * _[ i ]; 
	}
}

#include	<chrono>
using namespace chrono;

const size_t
BLOCK_SIZE	= CudaDeviceProp().maxThreadsPerBlock;

void
Main( size_t W ) {

	cerr << "numData : " << W / ( 1024 * 1024 ) << "MiB" << endl;
	cerr << "memory  : " << W * sizeof( FLOAT_TYPE ) * D / ( 1024 * 1024 * 1024 ) << "GiB" << endl;

	system_clock::time_point start;
	start = system_clock::now();

	FLOAT_TYPE queryHOST[ D ];
	for ( size_t _ = 0; _ < D; _++ ) queryHOST[ _ ] = (FLOAT_TYPE)( 1. / D );
	C( hipMemcpyToSymbol(HIP_SYMBOL( query), queryHOST, sizeof( FLOAT_TYPE ) * D ) );

	CUDAMemory< FLOAT_TYPE > data( W * D );
	DummyData( data  );
//data.DtoH();
//for ( size_t _ = 0; _ < W * D; _ += 1024 * 8 * D ) cerr << _ << ':' << (float)data( _ ) << endl;

	CUDAMemory< FLOAT_TYPE > result( W );

	cerr << "Initialize: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

	start = system_clock::now();
	DOT<<< ( W + BLOCK_SIZE ) / BLOCK_SIZE, BLOCK_SIZE >>>( data.$, result.$, W );
	result.DtoH(); //	hipDeviceSynchronize();
	cerr << "duration first: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

	start = system_clock::now();
	DOT<<< ( W + BLOCK_SIZE ) / BLOCK_SIZE, BLOCK_SIZE >>>( data.$, result.$, W );
	result.DtoH(); //	hipDeviceSynchronize();
	cerr << "duration second: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;

/*
	cerr << "STARING VALIDATION" << endl;
	start = system_clock::now();
	data.DtoH();
	auto p = data.Host();
	for ( size_t _ = 0; _ < W; _++ ) {
		auto q = p + D * _;
		double	$ = 0;
		for ( size_t i = 0; i < D; i++ ) $ += double( queryHOST[ i ] ) * double( q[ i ] );
		if ( abs( $ - (double)result( _ ) ) > 0.001 ) cout << _ << ':' << $ << ':' <<  (double)result( _ ) << endl;
	}
	cerr << "validation: " << duration_cast<milliseconds>( system_clock::now() - start ).count() << endl;
*/
}

int
main( int argc, char* argv[] ) {
	try {
		Main( size_t( 1024 * 1024 *  2 ) );
	} catch ( char* ex ) {
		cerr << ex << endl;
	}
}
